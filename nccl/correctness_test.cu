#include <iostream>
#include <vector>
#include <cmath>
#include <random>
#include <hip/hip_runtime.h>
#include <nccl.h>

#define CUDA_CHECK(cmd) do {                         \
  hipError_t err = cmd;                             \
  if (err != hipSuccess) {                          \
    std::cerr << "CUDA error: " << hipGetErrorString(err) \
              << " (" << __FILE__ << ":" << __LINE__ << ")\n"; \
    exit(EXIT_FAILURE);                              \
  }                                                  \
} while(0)

#define NCCL_CHECK(cmd) do {                         \
  ncclResult_t res = cmd;                            \
  if (res != ncclSuccess) {                          \
    std::cerr << "NCCL error: " << ncclGetErrorString(res) \
              << " (" << __FILE__ << ":" << __LINE__ << ")\n"; \
    exit(EXIT_FAILURE);                              \
  }                                                  \
} while(0)

// 用于比较浮点数是否在容差范围内
bool is_close(float a, float b, float rtol = 1e-5, float atol = 1e-8) {
    return std::fabs(a - b) <= (atol + rtol * std::fabs(b));
}

void test_broadcast(int size) {
    int nDevices = 0;
    CUDA_CHECK(hipGetDeviceCount(&nDevices));
    if (nDevices < 2) {
        std::cout << "需要至少2个GPU设备进行测试\n";
        return;
    }
    
    std::cout << "测试广播操作，数据大小: " << size << " 元素\n";
    
    // 为每个设备分配CUDA内存
    std::vector<float*> d_data(nDevices);
    std::vector<float*> h_results(nDevices);
    std::vector<hipStream_t> streams(nDevices);
    std::vector<int> devs(nDevices);
    ncclComm_t* comms = new ncclComm_t[nDevices];
    
    // 初始化数据
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    
    std::vector<float> h_data(size);
    for (int i = 0; i < size; i++) {
        h_data[i] = dist(gen);
    }
    
    // 初始化设备
    for (int i = 0; i < nDevices; i++) {
        devs[i] = i;
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipMalloc(&d_data[i], size * sizeof(float)));
        CUDA_CHECK(hipMemset(d_data[i], 0, size * sizeof(float)));
        h_results[i] = new float[size];
        CUDA_CHECK(hipStreamCreate(&streams[i]));
    }
    
    // 初始化NCCL
    NCCL_CHECK(ncclCommInitAll(comms, nDevices, devs.data()));
    
    // 只在root设备上设置数据
    int root = 0;
    CUDA_CHECK(hipSetDevice(root));
    CUDA_CHECK(hipMemcpy(d_data[root], h_data.data(), size * sizeof(float), hipMemcpyHostToDevice));
    
    // 执行广播
    NCCL_CHECK(ncclGroupStart());
    for (int i = 0; i < nDevices; i++) {
        CUDA_CHECK(hipSetDevice(i));
        NCCL_CHECK(ncclBroadcast(d_data[i], d_data[i], size, ncclFloat, root, comms[i], streams[i]));
    }
    NCCL_CHECK(ncclGroupEnd());
    
    // 同步并获取结果
    for (int i = 0; i < nDevices; i++) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipStreamSynchronize(streams[i]));
        CUDA_CHECK(hipMemcpy(h_results[i], d_data[i], size * sizeof(float), hipMemcpyDeviceToHost));
    }
    
    // 验证结果
    bool all_correct = true;
    for (int i = 1; i < nDevices; i++) {
        bool device_correct = true;
        for (int j = 0; j < size; j++) {
            if (!is_close(h_results[0][j], h_results[i][j])) {
                std::cout << "GPU " << i << " 数据不一致: index " << j 
                          << ", 值 " << h_results[i][j] 
                          << ", 预期 " << h_results[0][j] << "\n";
                device_correct = false;
                all_correct = false;
                break;
            }
        }
        if (device_correct) {
            std::cout << "GPU " << i << " 广播数据验证通过\n";
        }
    }
    
    if (all_correct) {
        std::cout << "广播操作验证通过: 所有GPU数据一致\n";
    } else {
        std::cout << "广播操作验证失败: 数据不一致\n";
    }
    
    // 清理资源
    for (int i = 0; i < nDevices; i++) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipFree(d_data[i]));
        delete[] h_results[i];
        CUDA_CHECK(hipStreamDestroy(streams[i]));
        ncclCommDestroy(comms[i]);
    }
    delete[] comms;
}

void test_allreduce(int size) {
    int nDevices = 0;
    CUDA_CHECK(hipGetDeviceCount(&nDevices));
    if (nDevices < 2) {
        std::cout << "需要至少2个GPU设备进行测试\n";
        return;
    }
    
    std::cout << "测试All-Reduce操作，数据大小: " << size << " 元素\n";
    
    // 为每个设备分配CUDA内存
    std::vector<float*> d_data(nDevices);
    std::vector<float*> h_input(nDevices);
    std::vector<float*> h_results(nDevices);
    std::vector<hipStream_t> streams(nDevices);
    std::vector<int> devs(nDevices);
    ncclComm_t* comms = new ncclComm_t[nDevices];
    
    // 初始化数据
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    
    for (int i = 0; i < nDevices; i++) {
        h_input[i] = new float[size];
        for (int j = 0; j < size; j++) {
            h_input[i][j] = dist(gen);
        }
    }
    
    // 初始化设备
    for (int i = 0; i < nDevices; i++) {
        devs[i] = i;
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipMalloc(&d_data[i], size * sizeof(float)));
        CUDA_CHECK(hipMemcpy(d_data[i], h_input[i], size * sizeof(float), hipMemcpyHostToDevice));
        h_results[i] = new float[size];
        CUDA_CHECK(hipStreamCreate(&streams[i]));
    }
    
    // 初始化NCCL
    NCCL_CHECK(ncclCommInitAll(comms, nDevices, devs.data()));
    
    // 执行All-Reduce (SUM)
    NCCL_CHECK(ncclGroupStart());
    for (int i = 0; i < nDevices; i++) {
        CUDA_CHECK(hipSetDevice(i));
        NCCL_CHECK(ncclAllReduce(d_data[i], d_data[i], size, ncclFloat, ncclSum, comms[i], streams[i]));
    }
    NCCL_CHECK(ncclGroupEnd());
    
    // 同步并获取结果
    for (int i = 0; i < nDevices; i++) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipStreamSynchronize(streams[i]));
        CUDA_CHECK(hipMemcpy(h_results[i], d_data[i], size * sizeof(float), hipMemcpyDeviceToHost));
    }
    
    // 计算预期结果
    std::vector<float> expected(size, 0.0f);
    for (int i = 0; i < nDevices; i++) {
        for (int j = 0; j < size; j++) {
            expected[j] += h_input[i][j];
        }
    }
    
    // 验证结果
    bool all_correct = true;
    for (int i = 0; i < nDevices; i++) {
        bool device_correct = true;
        for (int j = 0; j < size; j++) {
            if (!is_close(expected[j], h_results[i][j])) {
                std::cout << "GPU " << i << " 数据不一致: index " << j 
                          << ", 值 " << h_results[i][j] 
                          << ", 预期 " << expected[j] 
                          << ", 差值 " << std::fabs(h_results[i][j] - expected[j]) << "\n";
                device_correct = false;
                all_correct = false;
                if (j > 10) break; // 只显示前几个错误
            }
        }
        if (device_correct) {
            std::cout << "GPU " << i << " All-Reduce数据验证通过\n";
        }
    }
    
    if (all_correct) {
        std::cout << "All-Reduce操作验证通过: 所有GPU数据一致\n";
    } else {
        std::cout << "All-Reduce操作验证失败: 数据不一致\n";
    }
    
    // 清理资源
    for (int i = 0; i < nDevices; i++) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipFree(d_data[i]));
        delete[] h_input[i];
        delete[] h_results[i];
        CUDA_CHECK(hipStreamDestroy(streams[i]));
        ncclCommDestroy(comms[i]);
    }
    delete[] comms;
}

int main() {
    // 测试不同数据大小
    std::vector<int> sizes = {1024, 1024 * 1024, 10 * 1024 * 1024};
    
    for (int size : sizes) {
        test_broadcast(size);
        std::cout << "-------------------\n";
        test_allreduce(size);
        std::cout << "===================\n";
    }
    
    return 0;
}