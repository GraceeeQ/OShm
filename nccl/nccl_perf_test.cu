#include <iostream>
#include <vector>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>
#include <nccl.h>

#define CUDA_CHECK(cmd) do {                         \
  hipError_t err = cmd;                             \
  if (err != hipSuccess) {                          \
    std::cerr << "CUDA error: " << hipGetErrorString(err) \
              << " (" << __FILE__ << ":" << __LINE__ << ")\n"; \
    exit(EXIT_FAILURE);                              \
  }                                                  \
} while(0)

#define NCCL_CHECK(cmd) do {                         \
  ncclResult_t res = cmd;                            \
  if (res != ncclSuccess) {                          \
    std::cerr << "NCCL error: " << ncclGetErrorString(res) \
              << " (" << __FILE__ << ":" << __LINE__ << ")\n"; \
    exit(EXIT_FAILURE);                              \
  }                                                  \
} while(0)

// 测试一次通信操作的延迟
double measure_latency(ncclComm_t* comms, int nDevices, std::vector<float*>& d_data, 
                       std::vector<hipStream_t>& streams, int size, ncclRedOp_t op, 
                       ncclDataType_t dataType, int iterations) {
    // 热身
    NCCL_CHECK(ncclGroupStart());
    for (int i = 0; i < nDevices; i++) {
        CUDA_CHECK(hipSetDevice(i));
        NCCL_CHECK(ncclAllReduce(d_data[i], d_data[i], size, dataType, op, comms[i], streams[i]));
    }
    NCCL_CHECK(ncclGroupEnd());
    
    for (int i = 0; i < nDevices; i++) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipStreamSynchronize(streams[i]));
    }
    
    // 测量延迟
    auto start = std::chrono::high_resolution_clock::now();
    
    for (int iter = 0; iter < iterations; iter++) {
        NCCL_CHECK(ncclGroupStart());
        for (int i = 0; i < nDevices; i++) {
            CUDA_CHECK(hipSetDevice(i));
            NCCL_CHECK(ncclAllReduce(d_data[i], d_data[i], size, dataType, op, comms[i], streams[i]));
        }
        NCCL_CHECK(ncclGroupEnd());
        
        for (int i = 0; i < nDevices; i++) {
            CUDA_CHECK(hipSetDevice(i));
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
        }
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::micro> elapsed = end - start;
    
    return elapsed.count() / iterations;  // 单次延迟（微秒）
}

// 测试带宽
double measure_bandwidth(ncclComm_t* comms, int nDevices, std::vector<float*>& d_data, 
                         std::vector<hipStream_t>& streams, int size, ncclRedOp_t op, 
                         ncclDataType_t dataType, int iterations) {
    // 热身
    NCCL_CHECK(ncclGroupStart());
    for (int i = 0; i < nDevices; i++) {
        CUDA_CHECK(hipSetDevice(i));
        NCCL_CHECK(ncclAllReduce(d_data[i], d_data[i], size, dataType, op, comms[i], streams[i]));
    }
    NCCL_CHECK(ncclGroupEnd());
    
    for (int i = 0; i < nDevices; i++) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipStreamSynchronize(streams[i]));
    }
    
    // 计算带宽
    auto start = std::chrono::high_resolution_clock::now();
    
    for (int iter = 0; iter < iterations; iter++) {
        NCCL_CHECK(ncclGroupStart());
        for (int i = 0; i < nDevices; i++) {
            CUDA_CHECK(hipSetDevice(i));
            NCCL_CHECK(ncclAllReduce(d_data[i], d_data[i], size, dataType, op, comms[i], streams[i]));
        }
        NCCL_CHECK(ncclGroupEnd());
        
        for (int i = 0; i < nDevices; i++) {
            CUDA_CHECK(hipSetDevice(i));
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
        }
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    
    // 计算带宽: (数据大小 * 迭代次数 * 2(一次操作涉及的设备数)) / 时间
    // 对于AllReduce，每个GPU发送和接收 (n-1)*size/n 数据
    double bytes = size * sizeof(float) * iterations * 2 * (nDevices - 1) * nDevices / nDevices;
    return bytes / elapsed.count() / (1024 * 1024 * 1024);  // GB/s
}

void run_nccl_performance_tests() {
    int nDevices = 0;
    CUDA_CHECK(hipGetDeviceCount(&nDevices));
    if (nDevices < 2) {
        std::cout << "需要至少2个GPU设备进行测试\n";
        return;
    }
    
    std::cout << "运行NCCL性能测试 (使用 " << nDevices << " 个GPU)\n";
    
    // 测试不同的数据大小
    std::vector<int> sizes = {
        8, 64, 256, 1024,                                // 小规模 (字节)
        4 * 1024, 16 * 1024, 64 * 1024,                  // 中规模
        256 * 1024, 1024 * 1024,                         // 大规模
        4 * 1024 * 1024, 16 * 1024 * 1024                // 超大规模
    };
    
    std::vector<float*> d_data(nDevices);
    std::vector<hipStream_t> streams(nDevices);
    std::vector<int> devs(nDevices);
    ncclComm_t* comms = new ncclComm_t[nDevices];
    
    // 初始化设备
    for (int i = 0; i < nDevices; i++) {
        devs[i] = i;
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipStreamCreate(&streams[i]));
    }
    
    // 初始化NCCL
    NCCL_CHECK(ncclCommInitAll(comms, nDevices, devs.data()));
    
    // 为最大的数据大小分配内存
    int max_size = sizes.back();
    for (int i = 0; i < nDevices; i++) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipMalloc(&d_data[i], max_size * sizeof(float)));
        CUDA_CHECK(hipMemset(d_data[i], 0, max_size * sizeof(float)));
    }
    
    std::cout << "| 操作 | 大小(B) | 延迟(us) | 带宽(GB/s) | 吞吐量(GB/s) |\n";
    std::cout << "|------|---------|----------|------------|-------------|\n";
    
    // 测试不同数据大小的延迟和带宽
    for (int size_bytes : sizes) {
        int elements = size_bytes / sizeof(float);
        if (elements < 1) elements = 1;
        
        // 根据数据大小调整迭代次数
        int iterations = std::max(10, 10 * 1024 * 1024 / size_bytes);
        if (size_bytes > 1024 * 1024) iterations = std::max(10, iterations);
        
        // 测量AllReduce延迟
        double latency_us = measure_latency(comms, nDevices, d_data, streams, elements, ncclSum, ncclFloat, iterations);
        
        // 测量AllReduce带宽
        double bandwidth_gbs = measure_bandwidth(comms, nDevices, d_data, streams, elements, ncclSum, ncclFloat, iterations);
        
        // 计算吞吐量：数据大小/延迟
        double throughput_gbs = (size_bytes * 1e-9) / (latency_us * 1e-6);
        
        std::cout << "| AllReduce | " << size_bytes 
                  << " | " << latency_us 
                  << " | " << bandwidth_gbs 
                  << " | " << throughput_gbs << " |\n";
        
        // 测量Broadcast延迟
        latency_us = 0;
        bandwidth_gbs = 0;
        throughput_gbs = 0;
        
        // 热身
        NCCL_CHECK(ncclGroupStart());
        for (int i = 0; i < nDevices; i++) {
            CUDA_CHECK(hipSetDevice(i));
            NCCL_CHECK(ncclBroadcast(d_data[i], d_data[i], elements, ncclFloat, 0, comms[i], streams[i]));
        }
        NCCL_CHECK(ncclGroupEnd());
        
        for (int i = 0; i < nDevices; i++) {
            CUDA_CHECK(hipSetDevice(i));
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
        }
        
        // 测量延迟
        auto start = std::chrono::high_resolution_clock::now();
        
        for (int iter = 0; iter < iterations; iter++) {
            NCCL_CHECK(ncclGroupStart());
            for (int i = 0; i < nDevices; i++) {
                CUDA_CHECK(hipSetDevice(i));
                NCCL_CHECK(ncclBroadcast(d_data[i], d_data[i], elements, ncclFloat, 0, comms[i], streams[i]));
            }
            NCCL_CHECK(ncclGroupEnd());
            
            for (int i = 0; i < nDevices; i++) {
                CUDA_CHECK(hipSetDevice(i));
                CUDA_CHECK(hipStreamSynchronize(streams[i]));
            }
        }
        
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::micro> elapsed = end - start;
        latency_us = elapsed.count() / iterations;
        
        // 计算带宽和吞吐量
        std::chrono::duration<double> elapsed_s = std::chrono::duration_cast<std::chrono::duration<double>>(elapsed);
        double bytes = size_bytes * iterations * (nDevices - 1);  // root发送给其他n-1个设备
        bandwidth_gbs = bytes / elapsed_s.count() / (1024 * 1024 * 1024);
        throughput_gbs = (size_bytes * 1e-9) / (latency_us * 1e-6);
        
        std::cout << "| Broadcast | " << size_bytes 
                  << " | " << latency_us 
                  << " | " << bandwidth_gbs 
                  << " | " << throughput_gbs << " |\n";
    }
    
    // 清理资源
    for (int i = 0; i < nDevices; i++) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipFree(d_data[i]));
        CUDA_CHECK(hipStreamDestroy(streams[i]));
        ncclCommDestroy(comms[i]);
    }
    delete[] comms;
}

int main() {
    run_nccl_performance_tests();
    return 0;
}